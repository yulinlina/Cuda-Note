#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"
#include"error_check.h"
#include"time_helper.h"

// Todo
// Implement the cuda kernel function ***rgb_to_sepia_gpu***
__global__ void colorToGreyscaleConversion(unsigned char* g_output_image,unsigned char* g_input_image,int width,int channels){ 
     int col = threadIdx.x+blockIdx.x*blockDim.x;
     int row =threadIdx.y+blockIdx.y*blockDim.y;
     if (col<width && row < height){
    	 int offset = row*width + col;
    	 int rgboffset = offset*channels;
    	 unsigned char c1 = g_input_image[rgboffset]; // r通道的值
		 unsigned char c2 = g_input_image[rgboffset+1]; // g
		 unsigned char c3 = g_input_image[rgboffset+2];//r
        *(g_output_image + rgboffset) = (unsigned char)fmin((c1 * 0.393 + c2 * 0.769 + c3 * 0.189), 255.0);
        *(g_output_image + rgboffset + 1) = (unsigned char)fmin((c1 * 0.349 + c2 * 0.686 + c3 * 0.168), 255.0);
        *(g_output_image + rgboffset + 2) = (unsigned char)fmin((c1 * 0.272 + c2 * 0.534 + c3 * 0.131), 255.0);
        if (channels ==4){
                *(g_output_image + rgboffset + 3) = g_input_image[rgboffset + 3];
            }
     }
}

void rgb_to_sepia_cpu(unsigned char *input_image, unsigned char *output_image, int width, int height, int channels)
{
    for(int row=0; row<height; row++)
    {
        for(int col=0; col<width; col++)
        {
            int offset = (row*width + col)*channels;
            unsigned char c1 = input_image[offset];
            unsigned char c2 = input_image[offset+1];
            unsigned char c3 = input_image[offset+2];

            *(output_image + offset) = (unsigned char)fmin((c1 * 0.393 + c2 * 0.769 + c3 * 0.189), 255.0);
			*(output_image + offset + 1) = (unsigned char)fmin((c1 * 0.349 + c2 * 0.686 + c3 * 0.168), 255.0);
			*(output_image + offset + 2) = (unsigned char)fmin((c1 * 0.272 + c2 * 0.534 + c3 * 0.131), 255.0);

            if(channels==4)
            {
                *(output_image + offset + 3) = input_image[offset + 3];
            }
        }
    }
}


int main(int argc, char *argv[])
{
    if(argc<4)
    {
        printf("Usage: command    input-image-name    output-image-name option   option(cpu/gpu)");
        return -1;
    }
    char *input_image_name = argv[1];
    char *output_image_name = argv[2];
    char *option = argv[3];

    int width, height, original_no_channels;
    int desired_no_channels = 0; // Pass 0 to load the image as is
    unsigned char *stbi_img = stbi_load(input_image_name, &width, &height, &original_no_channels, desired_no_channels);
    if(stbi_img==NULL){ printf("Error in loading the image.\n"); exit(1);}
    printf("Loaded image with a width of %dpx, a height of %dpx. The original image had %d channels, the loaded image has %d channels.\n", width, height, original_no_channels, desired_no_channels);

    int channels = original_no_channels;
    int img_mem_size = width * height * channels * sizeof(char);
    double begin;
    if(strcmp(option, "cpu")==0)
    {
        printf("Processing with CPU!\n");
        unsigned char *sepia_img = (unsigned char *)malloc(img_mem_size);
        if(sepia_img==NULL){  printf("Unable to allocate memory for the sepia image. \n");  exit(1);  }

        
        // Time stamp
		begin = cpuSecond();

		// CPU computation (for reference)
		rgb_to_sepia_cpu(stbi_img, sepia_img, width, height, channels);

        // Time stamp
		printf("Time cost [CPU]:%f s\n", cpuSecond()-begin);

        // Save to an image file
        stbi_write_jpg(output_image_name, width, height, channels, sepia_img, 100);

        free(sepia_img);
    }
    else if(strcmp(option, "gpu")==0) 
    {
        printf("Processing with GPU!\n");

        //  Todo: 1. Allocate memory on GPU



        //  Todo: 2. Copy data from host memory to device memory



        //  Todo: 3. Call kernel function
        //        3.1 Declare block and grid sizes

        /*  dim3 block(..., ...);  
            dim3 grid(..., ...);  */

		//        3.2 Record the time cost of GPU computation
		begin = cpuSecond();

		//  Todo: 3.3 Call the kernel function (Don't forget to call hipDeviceSynchronize() before time recording)



		printf("Time cost [GPU]:%f s\n", cpuSecond()-begin);

		//  Todo:  4. Copy data from device to host



		//  Todo:  5. Save results as an image
        /*  stbi_write_jpg(output_image_name, width, height, channels, sepia_img_from_gpu, 100);  */



        //  Todo:  6. Release host memory and device memory



    } 
    else
    {
        printf("Unexpected option (please use cpu/gpu) !\n");
    }   

    stbi_image_free(stbi_img);

    return 0;
}