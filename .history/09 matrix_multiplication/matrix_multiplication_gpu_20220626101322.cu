#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include"matrix_helper.h"
#include"gpu_timer.h"

#define BLOCK_SIZE 16

__global__ void kernel_matrix_multiply(float *M, float *N, float *P, int M_rows, int M_cols, int N_rows, int N_cols)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    if(row<M_rows && col<N_cols)
    {
        float Pvalue = 0;
        for(int k=0; k<M_cols; k++){
            Pvalue += M[row*M_cols+k] * N[k*N_cols+col];
        }
        P[row*N_cols+col] = Pvalue;
    }       
}

/* 
* Todo:
* Implement the following block matrix multiplication kernel function *
*/
__global__ void kernel_matrix_multiply(float *M, float *N, float *P, int M_rows, int M_cols, int N_rows, int N_cols){
    __shared__ float Mds[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];
    int bx= blockIdx.x,by=blockIdx.y;
    int tx =threadIdx.x,ty=threadIdx.y;

    int row =by*BLOCK_SIZE+ty;
    int col=bx*BLOCK_SIZE+tx;

    float Pvalue=0;
    for(int ph=0;ph<(M_cols-1)/BLOCK_SIZE+1;ph++){
        if(row<M_rows &&ph*BLOCK_SIZE+tx<M_cols)
        Mds[ty][tx]=M[row*M_cols+ph*BLOCK_SIZE+tx];
        if((ph*BLOCK_SIZE+ty)<N_rows &&col<N_cols)
        Nds[ty][tx]=N[(ph*BLOCK_SIZE+ty)*N_cols+col];
        __syncthreads();

        for(int k=0;k<BLOCK_SIZE;k++){
            Pvalue+=Mds[ty][k]*Nds[k][tx];
        }
         __syncthreads();
    }
    if(row<M_rows&&col<N_cols)
    {
        P[row*N_cols+col]=Pvalue;
    }
}


void gpu_matrix_multiply(float *M, float *N, float *P, int M_rows, int M_cols, int N_rows, int N_cols,
   void(*kernel)(float *M, float *N, float *P, int M_rows, int M_cols, int N_rows, int N_cols) )
{
    float *d_M, *d_N, *d_P;
    hipMalloc((void **)&d_M, M_rows*M_cols*sizeof(float));
    hipMalloc((void **)&d_N, N_rows*N_cols*sizeof(float));
    hipMalloc((void **)&d_P, M_rows*N_cols*sizeof(float));
    
    hipMemcpy(d_M, M, M_rows*M_cols*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, N_rows*N_cols*sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_P, 0, M_rows*N_cols*sizeof(float));
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N_cols-1)/block.y+1, (M_rows-1)/block.x+1);
    // printf("\ngrid(%d, %d)\n", grid.x, grid.y);
    GpuTimer timer;
    timer.Start();
    kernel<<<grid, block>>>(d_M, d_N, d_P, M_rows, M_cols, N_rows, N_cols);
    timer.Stop();
    printf("Time cost:%f ms\n", timer.Elapsed());

    hipMemcpy(P, d_P, M_rows*N_cols*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}


int main(int argc, char **argv)
{   
    if(argc==1){
        int test_size = 1024;
        float *M1 = (float *)malloc(test_size * test_size * sizeof(float));
        float *N1 = (float *)malloc(test_size * test_size * sizeof(float));
        printf("Matrix Multiplication (GPU):\n");
        float *P1 = (float *)malloc(test_size * test_size * sizeof(float));
        float *Pb1 = (float *)malloc(test_size * test_size * sizeof(float));

        gpu_matrix_multiply(M1, N1, P1, test_size, test_size, test_size, test_size, kernel_matrix_multiply);
        gpu_matrix_multiply(M1, N1, Pb1, test_size, test_size, test_size, test_size, kernel_matrix_multiply_block);
        if(compare_matrix(P1, Pb1, test_size, test_size)==1){ printf("##Passed!\n\n"); }else{ printf("@@Failed!\n\n"); }
        free(M1);
        free(N1);
        free(P1);
        free(Pb1);
        exit(-1);
    }

    if(argc<4) {
        printf("Usage: command    matrix-M-filename    matrix-N-filename   matrix-P-filename   matrix-R-filename");
        return -1;
    }
    

    const char *matrix_M_filename = argv[1];
    const char *matrix_N_filename = argv[2];
    const char *matrix_P_filename = argv[3];
    const char *matrix_R_filename = argv[4];
    
    
    /* 
    *  Load M matrix *
    */
    int M_rows, M_cols;
    float *M = read_matrix(matrix_M_filename, &M_rows, &M_cols);
    
    /* 
    *  Load N matrix *
    */
    int N_rows, N_cols;
    float *N = read_matrix(matrix_N_filename, &N_rows, &N_cols);

    if(M_cols!=N_rows){
        printf("Rows and Cols don't match!");
        exit(-1);
    }
    
    /* 
    *  Matrix Multiplication  *
    */
    printf("Matrix Multiplication (GPU):\n");
    float *P = (float *)malloc(M_rows * N_cols * sizeof(float));

    gpu_matrix_multiply(M, N, P, M_rows, M_cols, N_rows, N_cols, kernel_matrix_multiply);
    

    printf("Block Matrix Multiplication (GPU):\n");
    float *Pb = (float *)malloc(M_rows * N_cols * sizeof(float));

    gpu_matrix_multiply(M, N, Pb, M_rows, M_cols, N_rows, N_cols, kernel_matrix_multiply_block);
    if(compare_matrix(P, Pb, M_rows, N_cols)==1){ printf("##Passed!\n\n"); }else{ printf("@@Failed!\n\n"); }
   /* 
    *  Save Results  *
    */
    write_matrix(matrix_P_filename, P, M_rows, N_cols);
    
    
    /* 
    *  Check Results  *
    */
    if(matrix_R_filename!=NULL){
        printf("\n Checking results ... \n");
        int R_rows, R_cols;
        float *R = read_matrix(matrix_R_filename, &R_rows, &R_cols);;
        if(compare_matrix(P, R, R_rows, R_cols)==1){ printf("##Passed!\n\n"); }else{ printf("@@Failed!\n\n"); }
        free(R);
    }
    
    free(M);
    free(N);
    free(P);
    free(Pb);
    
    return 0;
}
