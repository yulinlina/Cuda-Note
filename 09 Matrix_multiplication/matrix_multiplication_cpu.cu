#include<stdio.h>
#include<stdlib.h>
#include"matrix_helper.h"
#include"time_helper.h"

#define BLOCK_SIZE 16

void matrix_multiply(const float *M, const float *N, float *P, int M_rows, int M_cols, int N_rows, int N_cols)
{
    for(int row=0; row<M_rows; row++){
        for(int col=0; col<N_cols; col++){
            float Pvalue = 0;
            for(int k=0; k<M_cols; k++){
                Pvalue += M[row*M_cols+k] * N[k*N_cols+col];
            }
            P[row*N_cols+col] = Pvalue;
        }
    }
}

/* 
* Todo:
* Implement the following block matrix multiplication function *
*/
void matrix_multiply_block(const float *M, const float *N, float *P, int M_rows, int M_cols, int N_rows, int N_cols)
{

}

int main(int argc, char **argv)
{   
    double begin;

    if(argc==1){
        int test_size = 1024;
        float *M1 = (float *)malloc(test_size * test_size * sizeof(float));
        float *N1 = (float *)malloc(test_size * test_size * sizeof(float));
        printf("Matrix Multiplication (CPU):\n");
        float *P1 = (float *)malloc(test_size * test_size * sizeof(float));
        float *Pb1 = (float *)malloc(test_size * test_size * sizeof(float));

        begin = cpuSecond();
        matrix_multiply(M1, N1, P1, test_size, test_size, test_size, test_size);
        printf("Time cost:%lf s\n", cpuSecond()-begin);

        begin = cpuSecond();
        matrix_multiply_block(M1, N1, Pb1, test_size, test_size, test_size, test_size);
        printf("Time cost:%lf s\n", cpuSecond()-begin);

        if(compare_matrix(P1, Pb1, test_size, test_size)==1){ printf("##Passed!\n\n"); }else{ printf("@@Failed!\n\n"); }
        free(M1);
        free(N1);
        free(P1);
        free(Pb1);
        exit(-1);
    }

    if(argc<4) {
        printf("Usage: command    matrix-M-filename    matrix-N-filename   matrix-P-filename   matrix-R-filename");
        return -1;
    }
    const char *matrix_M_filename = argv[1];
    const char *matrix_N_filename = argv[2];
    const char *matrix_P_filename = argv[3];
    const char *matrix_R_filename = argv[4];
    
    
    /* 
    *  Load M matrix *
    */
    int M_rows, M_cols;
    float *M = read_matrix(matrix_M_filename, &M_rows, &M_cols);
    // print_matrix(M, M_rows, M_cols);
    
    
    /* 
    *  Load N matrix *
    */
    int N_rows, N_cols;
    float *N = read_matrix(matrix_N_filename, &N_rows, &N_cols);
    // print_matrix(N, N_rows, N_cols);

    if(M_cols!=N_rows){
        printf("Rows and Cols don't match!");
        exit(-1);
    }
    
    /* 
    *  Matrix Multiplication  *
    */
    printf("Matrix multiplication (CPU) \n");
    float *P = (float *)malloc(M_rows * N_cols * sizeof(float));
    begin = cpuSecond();
    matrix_multiply(M, N, P, M_rows, M_cols, N_rows, N_cols);
    printf("Time cost:%lf s\n", cpuSecond()-begin);
    

    /* 
    *  Block Matrix Multiplication  *
    */
    printf("Block Matrix Multiplication (CPU) \n");
    float *Pb = (float *)malloc(M_rows * N_cols * sizeof(float));
    begin = cpuSecond();
    matrix_multiply_block(M, N, Pb, M_rows, M_cols, N_rows, N_cols);
    printf("Time cost:%lf s\n", cpuSecond()-begin);

    if(compare_matrix(P, Pb, M_rows, N_cols)==1){ printf("Passed!\n"); }else{ printf("Failed!\n"); }


   /* 
    *  Save Results  *
    */
    write_matrix(matrix_P_filename, Pb, M_rows, N_cols);
    
    
    /* 
    *  Check Results  *
    */
    if(matrix_R_filename!=NULL){
        printf("\n Checking results ... \n");
        int R_rows, R_cols;
        float *R = read_matrix(matrix_R_filename, &R_rows, &R_cols);;
        if(compare_matrix(P, R, R_rows, R_cols)==1){ printf("Passed!\n"); }else{ printf("Failed!\n"); }
        if(compare_matrix(Pb, R, R_rows, R_cols)==1){ printf("Passed!\n"); }else{ printf("Failed!\n"); }
        free(R);
    }
    
    
    free(M);
    free(N);
    free(P);
    free(Pb);
    
    return 0;
}
