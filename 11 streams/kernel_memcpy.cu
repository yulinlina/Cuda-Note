#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "error_check.h"
#include "gpu_timer.h"

#define BLOCK_SIZE 128

typedef double DTYPE;

const int N = 30000; 
const int NUM_STREAMS = 30;

void vec_add_cpu(const DTYPE *h_x, const DTYPE *h_y, DTYPE *h_z, const int n)
{
    for (int i=0; i < n; i++)
    {
        h_z[i] = h_x[i] + h_y[i];
    }
}

int vec_compare(const DTYPE *h_x, const DTYPE *h_y, const int n)
{
    for(int x=0; x<n; x++){
        if(abs(h_x[x]-h_y[x])>1e-3){
            printf("Results don't match! [%d] [%f - %f]\n", x, h_y[x], h_x[x]);
            return -1;
        }            
    }
    return 1;
}

void __global__ vec_add_kernel(const DTYPE *d_x, const DTYPE *d_y, DTYPE *d_z, const int n)
{
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
    {
        // repeat to increase the computational cost
        for(int x=0; x<1000000; x++)
        {
                d_z[idx] = d_x[idx] + d_y[idx];
        }
    }
}

void vec_add_default_stream(const DTYPE *h_x, const DTYPE *h_y, DTYPE *h_z, const int n);

void vec_add_multiple_streams_overlapped(const DTYPE *h_x, const DTYPE *h_y, DTYPE *h_z, const int n);


int main(void)
{
    DTYPE *h_x, *h_y, *h_z;
    // Todo 1
    // Allocate host memory for pointers [*h_x, *h_y, *h_z] using hipHostMalloc


    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
        h_y[n] = 2.34;
    }
    
    vec_add_cpu(h_x, h_y, h_z, N);
    
    vec_add_default_stream(h_x, h_y, h_z, N);
        
    vec_add_multiple_streams_overlapped(h_x, h_y, h_z, N);


    // Todo 2
    // Free host memory pointers [*h_x, *h_y, *h_z] using hipHostFree


    return 0;
}


void vec_add_default_stream(const DTYPE *h_x, const DTYPE *h_y, DTYPE *h_z, const int n)
{
    DTYPE *d_x, *d_y, *d_z;
    DTYPE *h_z1 = (DTYPE*) malloc(sizeof(DTYPE) * N);
    
    CHECK(hipMalloc(&d_x, sizeof(DTYPE) * N));
    CHECK(hipMalloc(&d_y, sizeof(DTYPE) * N));
    CHECK(hipMalloc(&d_z, sizeof(DTYPE) * N));
    CHECK(hipMemcpy(d_x, h_x, sizeof(DTYPE) * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, sizeof(DTYPE) * N, hipMemcpyHostToDevice));

    const int grid_size = (n - 1) / BLOCK_SIZE + 1;
    
    GpuTimer timer;
    timer.Start();
    vec_add_kernel<<<grid_size, BLOCK_SIZE>>>(d_x, d_y, d_z, n);
    CHECK(hipMemcpy(h_z1, d_z, sizeof(DTYPE) * N, hipMemcpyDeviceToHost));
    timer.Stop();
    
    printf("[vec_add_default_stream] Time cost: %f ms\n", timer.Elapsed());     
    CHECK(hipDeviceSynchronize());
    if(vec_compare(h_z1, h_z, N)==1){ printf("  PASSED!\n");  }else{  printf("  FAILED\n");  }
    
    free(h_z1);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));    
}


// Todo 3
// Using multiple streams to tmplement the following function achieve overlapped memcpy [hipMemcpyAsync] and kernel computing
void vec_add_multiple_streams_overlapped(const DTYPE *h_x, const DTYPE *h_y, DTYPE *h_z, const int n)
{

}
